// From http://computer-graphics.se/hello-world-for-cuda.html


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

const int N = 16;
const int blocksize = 16;

__global__
void hello(char *a, int *b)
{
  a[threadIdx.x] += b[threadIdx.x];
}

int main()
{
  char a[N] = "Hello \0\0\0\0\0\0";
  int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

  char *ad;
  int *bd;
  const int csize = N*sizeof(char);
  const int isize = N*sizeof(int);

  printf("%s", a);

  hipMalloc( (void**)&ad, csize );
  hipMalloc( (void**)&bd, isize );
  hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
  hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
  hipFree( ad );
  hipFree( bd );

  assert(strcmp(a, "Hello World!"));

  return 0;
}